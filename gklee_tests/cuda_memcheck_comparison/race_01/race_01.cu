
#include <hip/hip_runtime.h>
#include <stdio.h>

#define SIZE 64
#define BLOCKS 1

__global__ void device_global(unsigned int *array_a, unsigned int *array_b, int num_elements) {
  int my_index = blockIdx.x * blockDim.x + threadIdx.x;

  __shared__ unsigned int my_shared[SIZE];
  my_shared[my_index] = my_index;
  __syncthreads();

  if (array_a[my_index] == array_b[my_index]) {
    if (array_b[my_index] == my_shared[my_index]) {
      array_a[0] = my_index;
    }
  } 
}

int main(int argc, char* argv[]) {
  // malloc
  unsigned int *host_array = (unsigned int*) malloc(SIZE*sizeof(unsigned int));
  unsigned int *device_array_a = 0;
  hipMalloc((void **) &device_array_a, SIZE*sizeof(unsigned int));
  unsigned int *device_array_b = 0;
  hipMalloc((void **) &device_array_b, SIZE*sizeof(unsigned int));

  // check malloc
  if (host_array == 0) { return 1;}
  if (device_array_a == 0) { return 2;}
  if (device_array_b == 0) { return 3;}

  // init array a to have 0 in odd indicies and 'index at even indicies
  for (int i=0; i<SIZE; i++) {
    if (i%2 == 0) {
      host_array[i] = i;
    } else {
      host_array[i] = 0;
    }
  }
  hipMemcpy(device_array_a, host_array, SIZE, hipMemcpyHostToDevice);

  // init array b to have 'index at indicies that are divisible by 2 and 3, 0 
  //     at other indicies
  for (int i=0; i<SIZE; i++) {
    if (i%2 == 0 && i%3 == 0) {
      host_array[i] = i;
    } else {
      host_array[i] = 0;
    }
  }
  hipMemcpy(device_array_b, host_array, SIZE, hipMemcpyHostToDevice);
  
  // run global and copy output
  device_global<<<BLOCKS,(SIZE/BLOCKS)>>> (device_array_a, device_array_b, SIZE);
  hipMemcpy(host_array, device_array_a, SIZE, hipMemcpyDeviceToHost);

  // print output
  for (int i=0; i<SIZE; i += SIZE/BLOCKS) {
    for (int j=0; j<SIZE/BLOCKS; j++){
      printf("%d, ", host_array[i+j]);
    }
    printf("\n");
  }
  
  // cleanup
  free(host_array);
  hipFree(device_array_a);
  hipFree(device_array_b);
}