
#include <hip/hip_runtime.h>
#include <stdio.h>
#define SIZE 64
#define BLOCKS 4

__global__ void device_global(unsigned int *array_a, int num_elements) {
  int my_index = blockIdx.x * blockDim.x + threadIdx.x;

  array_a[my_index] = my_index % (num_elements/2); // write overlaping indicies
  __syncthreads();

  int next_index = array_a[my_index]; // use written index 
  array_a[next_index] = my_index; // cause a write write race
}

int main(void) {
  // malloc arrays
  unsigned int *host_array = (unsigned int*) malloc(SIZE*sizeof(unsigned int));
  unsigned int *device_array_a = 0;
  hipMalloc((void **) &device_array_a, SIZE*sizeof(unsigned int));

  // check mallocs
  if (host_array == 0) { return 1;}
  if (device_array_a == 0) { return 2;}

  // init host array to 0
  for (int i=0; i<SIZE; i++) {
    host_array[i] = 0;
  }

  // copy to device, call global, and copy output back
  hipMemcpy(device_array_a, host_array, SIZE, hipMemcpyHostToDevice);
  device_global<<<BLOCKS,(SIZE/BLOCKS)>>> (device_array_a, SIZE);
  hipMemcpy(host_array, device_array_a, SIZE, hipMemcpyDeviceToHost);

  // print output
  for (int i=0; i<SIZE; i += SIZE/BLOCKS) {
    for (int j=0; j<SIZE/BLOCKS; j++){
      printf("%d, ", host_array[i+j]);
    }
    printf("\n");
  }

  // cleanup
  free(host_array);
  hipFree(device_array_a);
}